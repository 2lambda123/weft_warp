#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "warp_device.cuh"
#include "check_cuda.h"
#include "wfloat3.h"

__global__ void null_spatial_kernel(unsigned N, spatial_data* space){   

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;} 

	space[tid_in].x				= 0.0;
	space[tid_in].y				= 0.0;
	space[tid_in].z				= 0.0;
	space[tid_in].xhat			= 0.0;
	space[tid_in].yhat			= 0.0;
	space[tid_in].zhat			= 0.0;
	space[tid_in].surf_dist		= 10000.0;
	space[tid_in].enforce_BC	= 0;
	space[tid_in].norm[0]		= 1.0;
	space[tid_in].norm[1]		= 0.0;
	space[tid_in].norm[2]		= 0.0;

}

void null_spatial(unsigned NUM_THREADS, unsigned N, spatial_data* space){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	null_spatial_kernel <<< blks, NUM_THREADS >>> (  N, space );
	check_cuda(hipDeviceSynchronize());

}

