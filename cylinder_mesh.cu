#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "datadef.h"

using namespace optix;

rtBuffer<geom_data,1>               dims;
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned,  cellnum,     attribute cell_num, );
rtDeclareVariable(unsigned,  cellmat,     attribute cell_mat, );
rtDeclareVariable(unsigned,  cellfissile, attribute cell_fis, );
rtDeclareVariable(unsigned,  sense      , attribute cell_sense, );
rtDeclareVariable(float3,    normal,      attribute normal,   );

RT_PROGRAM void intersect(int object_dex)
{
	float t1, t2, sdisc, sgn;
	float3 this_norm1, this_norm2, int1, int2;

	float3 loc  = make_float3(dims[object_dex].loc[0],dims[object_dex].loc[1],dims[object_dex].loc[2]);
	float3 xformed_origin = ray.origin - loc;

	float radius    = dims[object_dex].max[0];
	float zmin 		= dims[object_dex].min[2]-loc.z;
	float zmax 		= dims[object_dex].max[2]-loc.z;

	float a =        ( ray.direction.x  * ray.direction.x  ) + ( ray.direction.y  * ray.direction.y  );
	float b = 2.0 * (( ray.direction.x  * xformed_origin.x ) + ( ray.direction.y  * xformed_origin.y ));
	float c =        ( xformed_origin.x * xformed_origin.x ) + ( xformed_origin.y * xformed_origin.y ) - (radius * radius);
	float disc = (b*b)-(4.0*a*c);

	bool report = false;
	bool check_second = true;
	sgn = 0.0;

	if (disc > 0.0f){  // the line intersects the circle

		report = true;

		sdisc = sqrt(disc);
		t1 = (-b-sdisc)/(2.0*a);
		t2 = (-b+sdisc)/(2.0*a);

		// intersection points
		int1 = ray.direction * t1 + xformed_origin;
		int2 = ray.direction * t2 + xformed_origin;

		// normals point to radius
		this_norm1 = make_float3(  int1.x/radius ,  int1.y/radius , 0 );
		this_norm2 = make_float3(  int2.x/radius ,  int2.y/radius , 0 );

		//miss in corners
		if( ((int1.z > zmax) & (int2.z > zmax)) | ((int1.z < zmin) & (int2.z < zmin)) ){  
			report=false;
		}
		else{   // t1 always smaller 

			if (int1.z > zmax ){  //  top intersection z1
				t1 = (zmax - xformed_origin.z) / ray.direction.z;
				this_norm1 = make_float3(0, 0, 1);
			}
			else if(int1.z < zmin ) { // bottom intersection z1
				t1 = (zmin - xformed_origin.z) / ray.direction.z;
				this_norm1 = make_float3(0, 0, -1);
			}

			if (int2.z > zmax){  //  top intersection z2
				t2 = (zmax - xformed_origin.z) / ray.direction.z;
				this_norm2 = make_float3(0, 0, 1);
			}
			else if(int2.z < zmin) { // bottom intersection z2
				t2 = (zmin - xformed_origin.z) / ray.direction.z;
				this_norm2 = make_float3(0, 0, -1);
			}
		}
	    
	}
	else if( (xformed_origin.x*xformed_origin.x+xformed_origin.y*xformed_origin.y)<(radius*radius) ) {  // exactly perpendicular

		report = true;

		t1 = (zmax - xformed_origin.z) / ray.direction.z;
		t2 = (zmin - xformed_origin.z) / ray.direction.z;

		// sense
		if (t1*t2 < 0.0 ){ // neg means inside
			sgn = -1.0;
		}
		else{
			sgn =  1.0;
		}
		
		// intersection points
		int1 = ray.direction * t1 + xformed_origin;
		int2 = ray.direction * t2 + xformed_origin;	

		// bottom always -1, top always +1, sense used to flip if point is inside
		this_norm1 = make_float3(0,0,1);
		this_norm2 = make_float3(0,0,-1);
	}

	// sense
	if (t1*t2 < 0.0 ){ // neg means inside
		sgn = -1.0;
	}
	else{
		sgn =  1.0;
	}

	// report
	if (report){
		if(t1>0){
			if (rtPotentialIntersection(t1)) {
				cellnum     = dims[object_dex].cellnum;
				cellmat     = dims[object_dex].matnum;
				cellfissile = dims[object_dex].is_fissile;
				normal 		= sgn*this_norm1;
				sense       = int(sgn);
				if(rtReportIntersection(0)){
					check_second=false;
				}
			}
		}
		if(check_second & t2>0){
			if (rtPotentialIntersection(t2)) {
				cellnum     = dims[object_dex].cellnum;
				cellmat     = dims[object_dex].matnum;
				cellfissile = dims[object_dex].is_fissile;
				normal 		= sgn*this_norm2;
				sense       = int(sgn);
				rtReportIntersection(0);
			}
		}
	}

}

RT_PROGRAM void bounds (int object_dex, float result[6])
{
	float3 mins = make_float3(-dims[object_dex].max[0],-dims[object_dex].max[0], dims[object_dex].min[2]);  //set all to the radius
  	float3 maxs = make_float3( dims[object_dex].max[0], dims[object_dex].max[0], dims[object_dex].max[2]);
  	float3 loc  = make_float3( dims[object_dex].loc[0], dims[object_dex].loc[1], dims[object_dex].loc[2]);
	
  	optix::Aabb* aabb = (optix::Aabb*)result;
  	aabb->set(mins+loc, maxs+loc);
}
