#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "datadef.h"

using namespace optix;

rtDeclareVariable(intersection_point, payload, rtPayload, ); 
rtDeclareVariable(float, int_dist, rtIntersectionDistance, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(unsigned,  cellnum,     attribute cell_num, );
rtDeclareVariable(unsigned,  cellmat,     attribute cell_mat, );
rtDeclareVariable(unsigned,  cellfissile, attribute cell_fis, );
rtDeclareVariable(unsigned,  sense      , attribute cell_sense, );
rtDeclareVariable(unsigned,  outer_cell,  , );
rtDeclareVariable(float3, normal, attribute normal, );

RT_PROGRAM void closest_hit()
{

	// stop ray iterations if outer cell is hit
	//if(cellnum==outer_cell){
	//	payload.cont=0;
	//}

	// always update current position and intersection distance, camera takes care of recording the first one
	payload.x=int_dist*ray.direction.x+ray.origin.x;
	payload.y=int_dist*ray.direction.y+ray.origin.y;
	payload.z=int_dist*ray.direction.z+ray.origin.z;
	payload.surf_dist = int_dist;
	//rtPrintf("closest normal %6.4E %6.4E %6.4E\n",normal.x,normal.y,normal.z);
	payload.norm[0] = normal.x;
	payload.norm[1] = normal.y;
	payload.norm[2] = normal.z;

	// update sense
	payload.sense += sense;

	//update mat, cell, fiss
	payload.mat  = cellmat;
	payload.cell = cellnum;
	payload.fiss = cellfissile;


}
