#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "warp_device.cuh"
#include "check_cuda.h"


__global__ void pop_fission_kernel(unsigned N, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_scanned){

	// get tid
	int tid = threadIdx.x+blockIdx.x*blockDim.x;

	// declare shared variables					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 
	__shared__	spatial_data*		space;	
	__shared__	float*				E;		
	__shared__	unsigned*			rn_bank;
	__shared__	unsigned*			yield;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		dist_scatter 				= d_xsdata[0].dist_scatter;
		dist_energy 				= d_xsdata[0].dist_energy; 
		space						= d_particles[0].space;
		E							= d_particles[0].E;
		rn_bank						= d_particles[0].rn_bank;
		yield						= d_particles[0].yield;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else (epecially returns)
	__syncthreads();

	// load history data
	unsigned		this_dex		=	index[    tid];
	float			this_E			=	E[        tid];
	unsigned		this_yield		=	yield[    tid];
	unsigned		rn				=	rn_bank[  tid];
	float			this_x			=	space[    tid].x;
	float			this_y			=	space[    tid].y;
	float			this_z			=	space[    tid].z;

	// get array position from prefix scan
	unsigned	position	=	d_scanned[tid];

	// make sure individual loads happen before anything else?
	__syncthreads();

	// return immediately if out of bounds
	if (tid >= N){return;}

	// check yield
	if (this_yield==0){
		return;
	}

	// another yield check
	if((d_scanned[tid+1]-d_scanned[tid]) == 0){
		printf("NOT RIGHT! \n");
		return;
	}

	// check E data pointers
	if(dist_energy == 0x0){
		printf("null pointer, energy array in continuum scatter!,tid %u\n",tid);
		return;
	}

	//constants
	const float  	pi			=   3.14159265359;

	// internal kernel variables
	float		nu_t0				=	0.0;
	float		nu_t1				=	0.0;
	float		nu_d0				=	0.0;
	float		nu_d1				=	0.0;
	float		beta				=	0.0;
	float		e0					=	0.0;
	float		e1					=	0.0;
	unsigned	data_dex 			=	0;
	float 		sampled_E			=	0.0;
	float 		phi, mu, E0, f, rn1;
	unsigned	this_law, this_len, this_intt, upper_len, lower_len, pre_index, pre_position;
	float		*this_var, *this_cdf, *this_pdf, *upper_var, *lower_var;

	// load dist info
	dist_data	this_edist, this_sdist;
	dist_data	sdist_lower	=	dist_scatter[this_dex].lower[0];
	dist_data	sdist_upper	=	dist_scatter[this_dex].upper[0];
	dist_data	edist_lower	=	dist_energy[ this_dex].lower[0];
	dist_data	edist_upper	=	dist_energy[ this_dex].upper[0];

	// copy nu values, energy points from dist, t is len, d is law
	memcpy(&nu_t0	, &sdist_lower.len, 1*sizeof(float));
	memcpy(&nu_t1	, &sdist_upper.len, 1*sizeof(float));
	memcpy(&nu_d0	, &sdist_lower.law, 1*sizeof(float));
	memcpy(&nu_d1	, &sdist_upper.law, 1*sizeof(float));
	memcpy(&e0		, &sdist_lower.erg, 1*sizeof(float));
	memcpy(&e1		, &sdist_upper.erg, 1*sizeof(float));

	// get interpolated beta value, beta = nu_d / nu_t
	beta	=	interpolate_linear_energy( this_E, e0, e1, nu_d0, nu_d1 ) / 
				interpolate_linear_energy( this_E, e0, e1, nu_t0, nu_t1 );

	// write new histories for this yield number
	for(unsigned k=0 ; k < this_yield ; k++ ){

		//get proper data index
		data_dex = position+k;

		// check if this neutron is delayed or prompt
		if ( get_rand(&rn) > beta ){

			// do individual stochastic mixing for this prompt neutron
			// pick upper or lower edist via stochastic mixing
			f	=	(this_E - edist_lower.erg) / (edist_upper.erg - edist_lower.erg);
			if( get_rand(&rn) > f ){
				this_edist	=	edist_lower;
			}
			else{
				this_edist	=	edist_upper;
			}

			// set pointers and parameters
			this_law	=	this_edist.law;
			this_len 	=	this_edist.len;
			this_intt	=	this_edist.intt;
			this_var	=	this_edist.var;
			this_cdf	=	this_edist.cdf;
			this_pdf	=	this_edist.pdf;
			upper_var	=	edist_upper.var;
			lower_var	=	edist_lower.var;
			upper_len	=	edist_upper.len;
			lower_len	=	edist_lower.len;

		}
		else{

			// pick upper or lower sdist (contains the delayed data) via stochastic mixing
			f	=	0.0;//(this_E - sdist_lower.var[0]) / (sdist_upper.erg - sdist_lower.erg);
			if( get_rand(&rn) > f ){
				this_sdist	=	sdist_lower;
			}
			else{
				this_sdist	=	sdist_upper;
			}

			// decode precursor intt, 100 place
			this_intt	=	(this_sdist.intt%1000-this_sdist.intt%100)/100;

			// decode precursor law, 1000 place
			this_law	=	 (this_sdist.intt%10000-this_sdist.intt%1000)/1000;

			// sample which precursor neutron is from
			rn1 = get_rand(&rn);
			for( pre_index=0; pre_index<6; pre_index++ ){ 
				if ( rn1 <= this_sdist.var[pre_index+1] ){
					break;
				}
			}

			// get position of data in vector and vector length
			pre_position	=	(unsigned) this_sdist.pdf[pre_index];  									// haha preposition...
			this_len		=	(unsigned) this_sdist.pdf[pre_index+1] - (unsigned) this_sdist.pdf[pre_index];

			// get pointers to sampled data
			this_var	=	&this_sdist.cdf[pre_position];
			this_cdf	=	&this_sdist.cdf[pre_position + ((unsigned)this_sdist.pdf[6])   ];   // last value in cdf if the total length of the combined 6-vectors
			this_pdf	=	&this_sdist.cdf[pre_position + ((unsigned)this_sdist.pdf[6])*2 ];
			upper_var	=	&this_sdist.cdf[pre_position];
			lower_var	=	&this_sdist.cdf[pre_position];
			upper_len	=	this_len;
			lower_len	=	this_len;

			printf("DELAYED this_E %6.4E f %6.4E  pre_index %u pre_position %u this_len %u this_var[0] %6.4E this_cdf[0] %6.4E this_pdf[0] %6.4E\n",this_E,f,pre_index,pre_position,this_len,this_var[0],this_cdf[0],this_pdf[0]);

		}
			
		// sample dist, passing the parameters/pointers to the sampled delayed/prompt emission data
		if (this_law ==4 ){
	
			// sample continuous tabular
			E0 = sample_continuous_tablular( 	this_len , 
												this_intt , 
												get_rand(&rn) , 
												this_var , 
												this_cdf, 
												this_pdf );
			//scale it to bins 
			sampled_E = scale_to_bins(	f, E0, 
										 this_var[0],  this_var[ this_len-1], 
										lower_var[0], lower_var[lower_len-1], 
										upper_var[0], upper_var[upper_len-1] );
		
			// check errors
			if (!isfinite(sampled_E) | sampled_E<=0.0){
				printf("Fission pop mis-sampled tid %i data_dex %u E %6.4E... setting to 2.5\n",tid,data_dex,sampled_E);
				sampled_E = 2.5;
			}
			
			// sample mu/phi isotropically
			mu  = 2.0*get_rand(&rn)-1.0;
			phi = 2.0*pi*get_rand(&rn);
	
		}
		else{
			printf("LAW %u NOT HANDLED IN FISSION POP!\n",this_law);
		}

		// set data
		E[     data_dex ] 				= sampled_E;
		space[ data_dex ].x				= this_x;
		space[ data_dex ].y				= this_y;
		space[ data_dex ].z				= this_z;
		space[ data_dex ].xhat			= sqrtf(1.0-(mu*mu))*cosf(phi);
		space[ data_dex ].yhat			= sqrtf(1.0-(mu*mu))*sinf(phi); 
		space[ data_dex ].zhat			= mu;
		space[ data_dex ].enforce_BC	= 0;
		space[ data_dex ].surf_dist		= 999999.0;
		
		//if(data_dex<=9){printf("array index %u, E = % 6.4E d_fissile_energy[ data_dex ] = % 6.4E\n",data_dex,sampled_E,E[ data_dex ]);}

	}

	// write current seed out
	rn_bank[tid] = rn;

}

void pop_fission( unsigned NUM_THREADS, unsigned N, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_scanned ){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	pop_fission_kernel <<< blks, NUM_THREADS >>> ( N, d_xsdata, d_particles, d_scanned);
	check_cuda(hipDeviceSynchronize());

}

