#include "hip/hip_runtime.h"
#include "optix.h"
#include <optix_world.h>
#include "datadef.h"

using namespace optix;

rtBuffer<source_point,1>            positions_buffer;
rtBuffer<unsigned,1>                rxn_buffer;
rtBuffer<unsigned,1>                remap_buffer;
rtBuffer<unsigned,1>                done_buffer;
rtBuffer<unsigned,1>                cellnum_buffer;
rtBuffer<unsigned,1>                matnum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint, launch_index_in, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );
rtDeclareVariable(unsigned,  outer_cell, , );
rtDeclareVariable(unsigned,  trace_type, , );
rtDeclareVariable(unsigned,  boundary_condition, , );

RT_PROGRAM void camera()
{
	//skip done particles

	//remap if 2
	unsigned launch_index;
	if(trace_type==2){
		launch_index=remap_buffer[launch_index_in];
		if(rxn_buffer[launch_index_in]>900){return;}
	}
	else{
		launch_index = launch_index_in;
	}

	// declare important stuff
<<<<<<< HEAD
	int                 sense = 0;
=======
	int                 sense;
>>>>>>> 3118af2c2a4c1900ecdbcb4a58f9cbf9a12d48e7
	float               epsilon=5.0e-4; 	
	intersection_point  payload;
	
	// init payload flags
	payload.sense = 0;
	payload.cell  = 999999;
	payload.mat   = 999999;
	payload.cell  = 999999;
	payload.fiss  = 0;
	
	// init ray
	float3 ray_direction  = make_float3(positions_buffer[launch_index].xhat, positions_buffer[launch_index].yhat, positions_buffer[launch_index].zhat);
	float3 ray_origin     = make_float3(positions_buffer[launch_index].x,    positions_buffer[launch_index].y,    positions_buffer[launch_index].z);
	optix::Ray ray        = optix::make_Ray( ray_origin, ray_direction, 0, epsilon, RT_DEFAULT_MAX );

	// first trace to find closest hit, set norm/distance, set bc flag
	rtTrace(top_object, ray, payload);
	sense = payload.sense;
	if(trace_type==2){
		positions_buffer[launch_index].surf_dist = payload.surf_dist; 
		positions_buffer[launch_index].norm[0]   = payload.norm[0];
		positions_buffer[launch_index].norm[1]   = payload.norm[1];
		positions_buffer[launch_index].norm[2]   = payload.norm[2];
		// write bc flag if first hit is outer cell
		if(payload.cell == outer_cell){
			positions_buffer[launch_index].enforce_BC = boundary_condition;
		}
		else{
			positions_buffer[launch_index].enforce_BC = 0;
		}
	}

	// find entering cell otherwise, trace will write, use downward z 
	ray_direction  = make_float3(0.0,0.0,-1.0);
	while(sense>=0 && payload.cell != outer_cell){
		ray_origin = make_float3(payload.x,payload.y,payload.z);
		ray = optix::make_Ray( ray_origin, ray_direction, 0, epsilon, RT_DEFAULT_MAX );
		rtTrace(top_object, ray, payload);
		sense += payload.sense;
	}

	// write cell/material numbers to buffer
	if(trace_type == 2){ //write material to buffer normally, write surface distance
		matnum_buffer[launch_index] 				= payload.mat;
		cellnum_buffer[launch_index] 				= payload.cell;
	}
	else if(trace_type == 3){  //write fissile flag if fissile query
		matnum_buffer[launch_index] 				= payload.fiss;
		cellnum_buffer[launch_index] 				= payload.cell;
		rxn_buffer[launch_index_in] 				= 818;
	}

}

RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf( "Caught exception 0x%X at launch index (%d)\n", code, launch_index_in);
	rtPrintExceptionDetails();
}
