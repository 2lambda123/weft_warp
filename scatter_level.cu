#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "wfloat3.h"
#include "binary_search.h"
#include "warp_device.cuh"

__global__ void iscatter_kernel(unsigned N, unsigned starting_index, unsigned* remap, unsigned* isonum, unsigned * index, unsigned * rn_bank, float * E, source_point * space, unsigned * rxn, float * awr_list, float * Q, unsigned * done, float** scatterdat, float** energydat){

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;}       

	// declare shared variables
	__shared__ 	unsigned			n_isotopes;				
	__shared__ 	unsigned			energy_grid_len;		
	__shared__ 	unsigned			total_reaction_channels;
	__shared__ 	unsigned*			rxn_numbers;			
	__shared__ 	unsigned*			rxn_numbers_total;		
	__shared__ 	float*				energy_grid;			
	__shared__ 	float*				rxn_Q;						
	__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 
	__shared__	spatial_data*		space;	
	__shared__	unsigned*			rxn;	
	__shared__	float*				E;		
	__shared__	float*				Q;		
	__shared__	unsigned*			rn_bank;
	__shared__	unsigned*			cellnum;
	__shared__	unsigned*			matnum;	
	__shared__	unsigned*			isonum;	
	__shared__	unsigned*			yield;	
	__shared__	float*				weight;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		n_isotopes					= d_xsdata[0].n_isotopes;								
		energy_grid_len				= d_xsdata[0].energy_grid_len;				
		total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		energy_grid 				= d_xsdata[0].energy_grid;						
		rxn_Q 						= d_xsdata[0].Q;												
		xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy; 
		space						= d_particles[0].space;
		rxn							= d_particles[0].rxn;
		E							= d_particles[0].E;
		Q							= d_particles[0].Q;	
		rn_bank						= d_particles[0].rn_bank;
		cellnum						= d_particles[0].cellnum;
		matnum						= d_particles[0].matnum;
		isonum						= d_particles[0].isonum;
		yield						= d_particles[0].yield;
		weight						= d_particles[0].weight;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	//remap to active
	int tid				=	remap[starting_index + tid_in];
	unsigned this_rxn 	=	rxn[  starting_index + tid_in];

	// print and return if wrong
	if ( this_rxn!=2 & (this_rxn < 51 | this_rxn > 90) ){printf("level scattering kernel accessing wrong reaction @ dex %u rxn %u\n",tid, this_rxn);return;} 

	//constants
	const float  pi           =   3.14159265359 ;
	const float  m_n          =   1.00866491600 ; // u
	const float  E_cutoff     =   1e-11;
	const float  E_max        =   20.0; //MeV

	// load history data
	wfloat3		hats_old(space[tid].xhat,space[tid].yhat,space[tid].zhat);
	unsigned	this_tope		=	isonum[  tid];
	unsigned	this_dex		=	index[   tid];
	float		this_E			=	E[       tid];
	float		this_Q			=	Q[       tid];
	unsigned	rn				=	rn_bank[ tid];
	float		this_awr		=	awr[this_tope];
	dist_data*	scatter_lower	=	dist_scatter[this_dex].lower;
	dist_data*	scatter_upper	=	dist_scatter[this_dex].upper;
	float		rn1				=	get_rand(&rn);

	// internal kernel variables
	float 		mu, phi, next_E, last_E;
    unsigned 	vlen, next_vlen, offset, k; 
    unsigned  	isdone = 0;
	float  		E_target     		=   0;
	float 		speed_target     	=   sqrtf(2.0*E_target/(this_awr*m_n));
	float  		speed_n          	=   sqrtf(2.0*this_E/m_n);
	float 		E_new				=   0.0;
	wfloat3 	v_n_cm,v_t_cm,v_n_lf,v_t_lf,v_cm, hats_new, hats_target, rotation_hat;
	float 		mu0,mu1,cdf0,cdf1,arg;
	float 		v_rel,E_rel;

	// ensure normalization
	hats_old = hats_old / hats_old.norm2();

	// make target isotropic
	mu				=	(2.0*   get_rand(&rn)) - 1.0;
	phi				=	 2.0*pi*get_rand(&rn);
	hats_target.x	=	sqrtf(1.0-(mu*mu))*cosf(phi);
	hats_target.y	=	sqrtf(1.0-(mu*mu))*sinf(phi); 
	hats_target.z	=	mu;

	//sample therm dist if low E
	if(this_E <= 600*kb*temp ){
		sample_therm(&rn,&mu,&speed_target,temp,this_E,this_awr);
		hats_target = hats_old.rotate(mu, get_rand(&rn));
	}
	else{
		speed_target = 0.0;
	}
	__syncthreads();
	
	// make speed vectors
	v_n_lf = hats_old    * speed_n;
	v_t_lf = hats_target * speed_target;

	// calculate  v_cm
	v_cm = (v_n_lf + (v_t_lf*this_awr))/(1.0+this_awr);

	//transform neutron velocity into CM frame
	v_n_cm = v_n_lf - v_cm;
	v_t_cm = v_t_lf - v_cm;
	
	// sample new phi, mu_cm
	phi = 2.0*pi*get_rand(&rn);
	if(this_Sarray == 0x0){
		mu= 2.0*rn1-1.0;       // assume CM isotropic scatter if null
	}
	else{  
		// sample the distribution, pick upper or lower via stochastic mixing

	}

	// pre rotation directions
	hats_old = v_n_cm / v_n_cm.norm2();
	hats_old = hats_old.rotate(mu, get_rand(&rn));

	// check arg to make sure not negative
	arg = v_n_cm.dot(v_n_cm) + 2.0*this_awr*this_Q/((this_awr+1.0)*m_n);
	if(arg < 0.0) { 
		arg=0.0;
	}
	v_n_cm = hats_old * sqrtf( arg );

	// transform back to L frame
	v_n_lf = v_n_cm + v_cm;
	hats_new = v_n_lf / v_n_lf.norm2();
	hats_new = hats_new / hats_new.norm2();  // get higher precision, make SURE vector is length one
	
	// calculate energy in L frame
	E_new = 0.5 * m_n * v_n_lf.dot(v_n_lf);

	// enforce limits
	if ( E_new <= E_cutoff | E_new > E_max ){
		isdone=1;
		this_rxn = 998;  // ecutoff code
		rxn[starting_index+tid_in] = this_rxn;
		printf("i CUTOFF, E = %10.8E\n",E_new);
	}

	// write results
	E[      tid]		= E_new;
	space[  tid].xhat	= hats_new.x;
	space[  tid].yhat	= hats_new.y;
	space[  tid].zhat	= hats_new.z;
	rn_bank[tid]		= rn;

}

void iscatter( hipStream_t stream, unsigned NUM_THREADS, unsigned N, unsigned starting_index, unsigned* remap, unsigned* isonum, unsigned * index, unsigned * rn_bank, float * E, source_point * space ,unsigned * rxn, float* awr_list, float * Q, unsigned* done, float** scatterdat, float** energydat){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	iscatter_kernel <<< blks, NUM_THREADS , 0 , stream >>> (  N, starting_index, remap, isonum, index, rn_bank, E, space, rxn, awr_list, Q, done, scatterdat, energydat);
	hipDeviceSynchronize();

}

