#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "wfloat3.h"
#include "LCRNG.cuh"
#include "binary_search.h"

__device__ void process_fission(unsigned this_yield, unsigned* rn, unsigned position, unsigned this_tope, unsigned this_awr, float this_E, source_point this_space, float* this_Earray, float* this_Sarray, source_point* space_out, float* E_out){

	// internal data
	unsigned 	k, n, offset, vlen, next_vlen, law, data_dex, intt;
	float 		sampled_E, phi, mu, rn1, rn2, last_E, next_E, e_start, E0, E1, Ek, next_e_start, next_e_end, last_e_start, last_e_end, diff, r;
	float 		cdf0, cdf1, e0, e1, m, pdf0, pdf1, arg,x,y,z,A, R;
	const float pi 		= 3.14159265359;
	const float Emin 	= 1e-11;
	const float Emax 	= 20.0;

	//read in values
	offset = 6;
	memcpy(&last_E,   	&this_Earray[0], sizeof(float));
	memcpy(&next_E,   	&this_Earray[1], sizeof(float));
	memcpy(&vlen,   	&this_Earray[2], sizeof(float));
	memcpy(&next_vlen,	&this_Earray[3], sizeof(float));
	memcpy(&law, 		&this_Earray[4], sizeof(float)); 
	memcpy(&intt, 		&this_Earray[5], sizeof(float)); 


	if (law == 4){
		r = (this_E-last_E)/(next_E-last_E);
		last_e_start = this_Earray[ offset ];
		last_e_end   = this_Earray[ offset + vlen - 1 ];
		next_e_start = this_Earray[ offset + 3*vlen ];
		next_e_end   = this_Earray[ offset + 3*vlen + next_vlen - 1];
	}

	// loop over the for the number of (rebased) yielded particles
	for(k=0 ; k < this_yield ; k++ ){

		//get proper data index
		data_dex = position+k ;
		
		if (law==4){ // tabular
			rn1 = get_rand(rn);
			rn2 = get_rand(rn);
	
			//sample energy dist
			sampled_E = 0.0;
			if(  rn2 >= r ){   //sample last E
				diff = next_e_end - next_e_start;
				e_start = next_e_start;
				for ( n=0 ; n<vlen-1 ; n++ ){
					cdf0 		= this_Earray[ (offset +   vlen ) + n+0];
					cdf1 		= this_Earray[ (offset +   vlen ) + n+1];
					pdf0		= this_Earray[ (offset + 2*vlen ) + n+0];
					pdf1		= this_Earray[ (offset + 2*vlen ) + n+1];
					e0  		= this_Earray[ (offset          ) + n+0];
					e1  		= this_Earray[ (offset          ) + n+1]; 
					if( rn1 >= cdf0 & rn1 < cdf1 ){
						break;
					}
				}
			}
			else{
				diff = next_e_end - next_e_start;
				e_start = next_e_start;
				for ( n=0 ; n<next_vlen-1 ; n++ ){
					cdf0 		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+0];
					cdf1  		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+1];
					pdf0		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+0];
					pdf1		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+1];
					e0   		= this_Earray[ (offset + 3*vlen               ) + n+0];
					e1   		= this_Earray[ (offset + 3*vlen               ) + n+1];
					if( rn1 >= cdf0 & rn1 < cdf1 ){
						break;
					}
				}
			}
		
			if (intt==2){// lin-lin interpolation
				m 	= (pdf1 - pdf0)/(e1-e0);
				arg = pdf0*pdf0 + 2.0 * m * (rn1-cdf0);
				if(arg<0){
					E0 = e0 + (e1-e0)/(cdf1-cdf0)*(rn1-cdf0);
				}
				else{
					E0 	= e0 + (  sqrtf( arg ) - pdf0) / m ;
				}
			}
			else if(intt==1){// histogram interpolation
				E0 = e0 + (rn1-cdf0)/pdf0;
			}
			
			//scale it
			E1 = last_e_start + r*( next_e_start - last_e_start );
			Ek = last_e_end   + r*( next_e_end   - last_e_end   );
			sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;

		}
		else if (law==44){
			if(this_Sarray == 0x0){  //Sarray value is nu for 918, might have to put check in here
				printf("null pointer in pop multiplicity Sarray!, tope %u E %6.4E\n",this_tope,this_E);
				return;
			}
			//printf("%6.4E %6.4E %u %u %u ... %6.4E %6.4E ... %6.4E %6.4E %6.4E\n",this_Earray[0],this_Earray[1],vlen,next_vlen,law,this_Earray[5],this_Earray[6],this_Earray[offset],this_Earray[offset+vlen],this_Earray[offset+vlen+1]);
			if(this_E<last_E){this_E=last_E;}
			float r = (this_E-last_E)/(next_E-last_E);
			if(r<0){
				printf("r less than zero in source pop for multiplicity, r % 10.8E isotope %u this_E % 10.8E last_E % 10.8E next_E % 10.8E\n",r,this_tope,this_E,last_E,next_E);
			}
			last_e_start = this_Earray[ offset ];
			last_e_end   = this_Earray[ offset + vlen - 1 ];
			next_e_start = this_Earray[ offset + 3*vlen ];
			next_e_end   = this_Earray[ offset + 3*vlen + next_vlen - 1];
	
			//sample energy dist
			sampled_E = 0.0;
			rn1 = get_rand(rn);
			if(  get_rand(rn) >= r ){   //sample last E
				diff = last_e_end - last_e_start;
				e_start = last_e_start;
				//n = binary_search( &this_Earray[ offset + vlen ] , rn1, vlen);
				for ( n=0 ; n<vlen-1 ; n++ ){
					cdf0 		= this_Earray[ (offset +   vlen ) + n+0];
					cdf1 		= this_Earray[ (offset +   vlen ) + n+1];
					pdf0		= this_Earray[ (offset + 2*vlen ) + n+0];
					pdf1		= this_Earray[ (offset + 2*vlen ) + n+1];
					e0  		= this_Earray[ (offset          ) + n+0];
					e1  		= this_Earray[ (offset          ) + n+1]; 
					if( rn1 >= cdf0 & rn1 < cdf1 ){
						break;
					}
				}
				offset = 6;
				A = this_Sarray[ (offset)      + n ];
				R = this_Sarray[ (offset+vlen) + n ];
			}
			else{
				diff = next_e_end - next_e_start;
				e_start = next_e_start;
				//n = binary_search( &this_Earray[ offset + 3*vlen + next_vlen] , rn1, next_vlen);
				for ( n=0 ; n<next_vlen-1 ; n++ ){
					cdf0 		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+0];
					cdf1  		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+1];
					pdf0		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+0];
					pdf1		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+1];
					e0   		= this_Earray[ (offset + 3*vlen               ) + n+0];
					e1   		= this_Earray[ (offset + 3*vlen               ) + n+1];
					if( rn1 >= cdf0 & rn1 < cdf1 ){
						break;
					}
				}
				offset = 6;
				A = this_Sarray[ (offset+3*vlen)           +n  ] ;
				R = this_Sarray[ (offset+3*vlen+next_vlen) +n  ];
			}
			
			// histogram interpolation, intt=1
			E0 = e0 + (rn1-cdf0)/pdf0;
			
			//scale it
			E1 = last_e_start + r*( next_e_start - last_e_start );
			Ek = last_e_end   + r*( next_e_end   - last_e_end   );
			sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;
			
			// find mu
			rn1 = get_rand(rn);
			if(get_rand(rn)>R){
				float T = (2.0*rn1-1.0)*sinhf(A);
				mu = logf(T+sqrtf(T*T+1.0))/A;
			}
			else{
					mu = logf(rn1*expf(A)+(1.0-rn1)*expf(-A))/A;
				}
		}
		else if(law==7){   // maxwellian fission

			// get tabulated temperature
			float t0 = this_Earray[ offset     ];
			float t1 = this_Earray[ offset + 1 ];
			float U  = this_Earray[ offset + vlen       ];
			      e0 = this_Earray[ offset + vlen*2     ];
			      e1 = this_Earray[ offset + vlen*2 + 1 ];
			float  T = 0.0;

			// interpolate T
			if (e1==e0){  // in top bin, both values are the same
				T = t0;
			}
			else if (intt==2){// lin-lin interpolation
				m = (this_E - e0)/(e1 - e0);
                T = (1.0 - m)*t0 + m*t1;
			}
			else if(intt==1){// histogram interpolation
				T  = (t1 - t0)/(e1 - e0) * this_E + t0;
			}
			else{
				printf("something is wrong in pop for law 7");
			}

			// rejection sample
			sampled_E = 9999999.9;
			while ( sampled_E > (this_E - U)){
				m = cosf(pi*get_rand(rn)/2.0);
				sampled_E = -T * ( m*m*logf(get_rand(rn))  +   logf(get_rand(rn)) );
			}

		}
		else if (law==9){   //evaporation spectrum

			// get tabulated temperature  
			float t0 = this_Earray[ offset              ];
			float t1 = this_Earray[ offset + 1          ];
			float U  = this_Earray[ offset + vlen       ];
			      e0 = this_Earray[ offset + vlen*2     ];
			      e1 = this_Earray[ offset + vlen*2 + 1 ];
			float  T = 0.0;

			// interpolate T
			if (e1==e0){  // in top bin, both values are the same
				T = t0;
			}
			else if (intt==2){// lin-lin interpolation
				m = (this_E - e0)/(e1 - e0);
                T = (1.0 - m)*t0 + m*t1;
			}
			else if(intt==1){// histogram interpolation
				T  = (t1 - t0)/(e1 - e0) * this_E + t0;
			}

			// rejection sample
			m  = (this_E - U)/T;
			e0 = 1.0-expf(-m);
			x  = -logf(1.0-e0*get_rand(rn)) - logf(1.0-e0*get_rand(rn));
			while (  x>m ) {
				x  = -logf(1.0-e0*get_rand(rn)) - logf(1.0-e0*get_rand(rn));
			}

			// mcnp5 volIII pg 2-43
			sampled_E = T * x;

			//isotropic mu
			mu  = 2.0*get_rand(rn)-1.0;

		}
		else{
			printf("LAW %u NOT HANDLED IN POP!\n",law);
		}

		//sample isotropic directions
		rn1 = get_rand(rn);
		rn2 = get_rand(rn);
		mu  = 2.0*rn1-1.0; 
		phi = 2.0*pi*rn2;
		x = sqrtf(1.0-(mu*mu))*cosf(phi);
		y = sqrtf(1.0-(mu*mu))*sinf(phi);
		z = mu;
		//printf("% 6.4E % 6.4E % 6.4E\n",x,y,z);
	
		//check limits
		if (sampled_E >= Emax){sampled_E = Emax * 0.99;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}
		if (sampled_E <= Emin){sampled_E = Emin * 1.01;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}

		// sync before writes
		__syncthreads();

		// set data
		//printf("(xyz) %6.4E %6.4E %6.4E (dir) %6.4E %6.4E %6.4E E %6.4E\n",this_space.x,this_space.y,this_space.z,this_space.xhat,this_space.yhat,this_space.zhat,sampled_E);
		space_out[ data_dex ].x 			= this_space.x;
		space_out[ data_dex ].y 			= this_space.y;
		space_out[ data_dex ].z 			= this_space.z;
		space_out[ data_dex ].xhat 			= x;
		space_out[ data_dex ].yhat 			= y;
		space_out[ data_dex ].zhat 			= z;
		space_out[ data_dex ].enforce_BC 	= 0;
		space_out[ data_dex ].surf_dist 	= 99999.0;
		space_out[ data_dex ].macro_t 		= 8.675309;
		E_out 	 [ data_dex ] 				= sampled_E;
	}

}
//__device__ void process_multiplicity(unsigned this_yield, unsigned* rn, unsigned position, unsigned this_tope, unsigned this_awr, float this_E, source_point this_space, float* this_Earray, float* this_Sarray, source_point* space_out, float* E_out){
//
//	//constants
//	//const float  pi           =   3.14159265359 ;
//	const float  m_n  =   1.00866491600 ; // u
//	const float  Emin =   1e-11;
//	const float  Emax =   20.0; //MeV
//
//	// internal kernel variables
//	wfloat3 	hats_old(this_space.xhat,this_space.yhat,this_space.zhat);
//	float 		mu, next_E, last_E, sampled_E, e_start, E0, E1, Ek, next_e_start, next_e_end, last_e_start, last_e_end, diff;
//    unsigned 	k, vlen, next_vlen, offset, n, law, data_dex, intt; 
//	float  		speed_n          	=   sqrtf(2.0*this_E/m_n);
//	wfloat3 	v_n_cm,v_t_cm,v_n_lf,v_t_lf,v_cm, hats_new, hats_target;
//	float 		cdf0,e0,A,R,pdf0,rn1,cdf1,pdf1,e1;
//
//	//get proper data index
//	data_dex = position+k ;
//	
//	// make speed vectors
//	v_n_lf = hats_old    * speed_n;
//	v_t_lf = hats_target * 0.0;
//
//	// calculate  v_cm
//	v_cm = (v_n_lf + (v_t_lf*this_awr))/(1.0+this_awr);
//
//	//transform neutron velocity into CM frame
//	v_n_cm = v_n_lf - v_cm;
//	v_t_cm = v_t_lf - v_cm;
//
//	//
//	//sample energy
//	//
//	//read in values
//	offset = 6;
//	memcpy(&last_E,   	&this_Earray[0], sizeof(float));
//	memcpy(&next_E,   	&this_Earray[1], sizeof(float));
//	memcpy(&vlen,   	&this_Earray[2], sizeof(float));
//	memcpy(&next_vlen,	&this_Earray[3], sizeof(float));
//	memcpy(&law, 		&this_Earray[4], sizeof(float));
//	memcpy(&intt, 		&this_Earray[5], sizeof(float));
//
//	if (law==4){
//		float r = (this_E-last_E)/(next_E-last_E);
//		last_e_start = this_Earray[ offset ];
//		last_e_end   = this_Earray[ offset + vlen - 1 ];
//		next_e_start = this_Earray[ offset + 3*vlen ];
//		next_e_end   = this_Earray[ offset + 3*vlen + next_vlen - 1];
//		rn1 = get_rand(rn);
//		float rn2 = get_rand(rn);
//	
//		//sample energy dist
//		sampled_E = 0.0;
//		if(  rn2 >= r ){   //sample last E
//			diff = next_e_end - next_e_start;
//			e_start = next_e_start;
//			for ( n=0 ; n<vlen-1 ; n++ ){
//				cdf0 		= this_Earray[ (offset +   vlen ) + n+0];
//				cdf1 		= this_Earray[ (offset +   vlen ) + n+1];
//				pdf0		= this_Earray[ (offset + 2*vlen ) + n+0];
//				pdf1		= this_Earray[ (offset + 2*vlen ) + n+1];
//				e0  		= this_Earray[ (offset          ) + n+0];
//				e1  		= this_Earray[ (offset          ) + n+1]; 
//				if( rn1 >= cdf0 & rn1 < cdf1 ){
//					break;
//				}
//			}
//		}
//		else{
//			diff = next_e_end - next_e_start;
//			e_start = next_e_start;
//			for ( n=0 ; n<next_vlen-1 ; n++ ){
//				cdf0 		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+0];
//				cdf1  		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+1];
//				pdf0		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+0];
//				pdf1		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+1];
//				e0   		= this_Earray[ (offset + 3*vlen               ) + n+0];
//				e1   		= this_Earray[ (offset + 3*vlen               ) + n+1];
//				if( rn1 >= cdf0 & rn1 < cdf1 ){
//					break;
//				}
//			}
//		}
//		
//		if (intt==2){// lin-lin interpolation
//			float m 	= (pdf1 - pdf0)/(e1-e0);
//			float arg = pdf0*pdf0 + 2.0 * m * (rn1-cdf0);
//			if(arg<0){
//				E0 = e0 + (e1-e0)/(cdf1-cdf0)*(rn1-cdf0);
//			}
//			else{
//				E0 	= e0 + (  sqrtf( arg ) - pdf0) / m ;
//			}
//		}
//		else if(intt==1){// histogram interpolation
//			E0 = e0 + (rn1-cdf0)/pdf0;
//		}
//		
//		//scale it
//		E1 = last_e_start + r*( next_e_start - last_e_start );
//		Ek = last_e_end   + r*( next_e_end   - last_e_end   );
//		sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;
//
//		//isotropic mu
//		mu  = 2.0*get_rand(rn)-1.0;
//
//	}	
//
//	else if (law==9){   //evaopration spectrum
//
//		// get tabulated temperature
//		float t0 = this_Earray[ offset              ];
//		float t1 = this_Earray[ offset + 1          ];
//		float U  = this_Earray[ offset + vlen       ];
//		      e0 = this_Earray[ offset + vlen*2     ];
//		      e1 = this_Earray[ offset + vlen*2 + 1 ];
//		float  T = 0.0;
//		float  m = 0.0;
//
//		// interpolate T
//		if (e1==e0){  // in top bin, both values are the same
//				T = t0;
//			}
//		else if (intt==2){// lin-lin interpolation
//			m = (this_E - e0)/(e1 - e0);
//            T = (1.0 - m)*t0 + m*t1;
//		}
//		else if(intt==1){// histogram interpolation
//			T  = (t1 - t0)/(e1 - e0) * this_E + t0;
//		}
//
//		// rejection sample
//		m  = (this_E - U)/T;
//		e0 = 1.0-expf(-m);
//		float x  = -logf(1.0-e0*get_rand(rn)) - logf(1.0-e0*get_rand(rn));
//		while (  x>m ) {
//			x  = -logf(1.0-e0*get_rand(rn)) - logf(1.0-e0*get_rand(rn));
//		}
//
//		// mcnp5 volIII pg 2-43
//		sampled_E = T * x;
//
//		//isotropic mu
//		mu  = 2.0*get_rand(rn)-1.0;
//
//	}
//	else{
//		printf("LAW %u NOT HANDLED IN multiplicity POP!\n",law);
//	}
//		
//	// rotate direction vector
//	hats_old = v_n_cm / v_n_cm.norm2();
//	hats_old = hats_old.rotate(mu, get_rand(rn));
//	
//	//  scale to sampled energy
//	v_n_cm = hats_old * sqrtf(2.0*sampled_E/m_n);
//	
//	// transform back to L
//	v_n_lf = v_n_cm + v_cm;
//	hats_new = v_n_lf / v_n_lf.norm2();
//	hats_new = hats_new / hats_new.norm2(); // get higher precision, make SURE vector is length one
//	
//	// calculate energy in lab frame
//	//sampled_E = 0.5 * m_n * v_n_lf.dot(v_n_lf);
//
//	//check limits
//	if (sampled_E >= Emax){sampled_E = Emax * 0.99;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}
//	if (sampled_E <= Emin){sampled_E = Emin * 1.01;}//printf("enforcing limits in pop data_dex=%u, sampled_E = %6.4E\n",data_dex,sampled_E);}
//
//	// sync before writes
//	__syncthreads();
//
//	// write results
//	space_out[ data_dex ].x 			= this_space.x;
//	space_out[ data_dex ].y 			= this_space.y;
//	space_out[ data_dex ].z 			= this_space.z;
//	space_out[ data_dex ].xhat 			= hats_new.x;
//	space_out[ data_dex ].yhat 			= hats_new.y;
//	space_out[ data_dex ].zhat 			= hats_new.z;
//	space_out[ data_dex ].enforce_BC 	= 0;
//	space_out[ data_dex ].surf_dist 	= 99999.0;
//	space_out[ data_dex ].macro_t 		= 8.675309;
//	E_out 	 [ data_dex ] 				= sampled_E;
//
//}
__global__ void pop_source_kernel(unsigned N, unsigned* isonum, unsigned* completed, unsigned* scanned, unsigned* remap, unsigned* yield, unsigned* done, unsigned* index, unsigned* rxn, source_point* space, float* E , unsigned* rn_bank, float**  energydata, float**  scatterdata, source_point* space_out, float* E_out, float * awr_list){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}

	// return if no yield
	unsigned 		this_yield 	= yield[tid];
	if (this_yield==0){return;}

	// load in external data
	source_point 	this_space 	= space  [tid];
	unsigned 		position 	= scanned[tid];
	unsigned 		this_tope 	= isonum [tid];
	unsigned 		dex  		= index  [tid];
	unsigned 		rn 			= rn_bank[tid];
	unsigned 		this_rxn 	= rxn    [tid];
	float 			this_E 		= E      [tid]; 

	__syncthreads();

	float*	 		this_Sarray = scatterdata[dex];
	float*			this_Earray = energydata [dex];

	__syncthreads();

	// check data array pointers
	if(this_Earray == 0x0){
		printf("null pointer in pop Earray,tid %u dex %u rxn %u tope %u E %6.4E\n",tid,dex,this_rxn,this_tope,this_E);
		return;
	}
	//if(this_Sarray == 0x0){  //Sarray value is nu for 918, might have to put check in here
	//	printf("null pointer in pop Sarray!,tid %u dex %u rxn %u tope %u E %6.4E\n",tid,dex,this_rxn,this_tope,this_E);
	//	return;
	//}

	// sampled based on reaction type
	if(this_rxn>=916 & this_rxn<=945 ){
		     process_fission(this_yield, &rn, position, this_tope, awr_list[this_tope], this_E, this_space, this_Earray, this_Sarray,  space_out, E_out);
	//}
	//else if(this_rxn == 916 | this_rxn==924 | this_rxn == 911 | this_rxn == 924 | this_rxn == 929 | this_rxn == 930 | this_rxn == 941 | this_rxn == 917 | this_rxn == 925 | this_rxn == 942 ){
	//else if (){
		//printf("processing multiplicity rxn %d\n",this_rxn);
	//	process_multiplicity(this_yield, &rn, position, this_tope, awr_list[this_tope], this_E, this_space, this_Earray, this_Sarray, space_out, E_out);
	}
	else{
		printf("tid %u REACTION %u HAS NONZERO YIELD IN SOURCE POP!\n",tid,this_rxn);
		return;
	}

	// update rn bank
	rn_bank[tid] = rn;

}

void pop_source( unsigned NUM_THREADS,  unsigned N, unsigned* isonum, unsigned* d_completed, unsigned* d_scanned, unsigned* d_remap, unsigned* d_yield, unsigned* d_done, unsigned* d_index, unsigned* d_rxn, source_point* d_space, float* d_E , unsigned* d_rn_bank, float ** energydata, float** scatterdata, source_point* space_out, float* E_out, float * awr_list){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	pop_source_kernel <<< blks, NUM_THREADS >>> ( N, isonum, d_completed, d_scanned, d_remap, d_yield, d_done, d_index, d_rxn, d_space, d_E , d_rn_bank, energydata, scatterdata, space_out, E_out, awr_list);
	hipDeviceSynchronize();

}

