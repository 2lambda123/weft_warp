#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "wfloat3.h"
#include "warp_device.cuh"

__global__ void fission_kernel(unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;}       

	// declare shared variables
	__shared__ 	dist_container*		dist_scatter;			
	__shared__	unsigned*			rxn;	
	__shared__	unsigned*			rn_bank;
	__shared__	unsigned*			yield;	
	__shared__	float*				weight;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){											
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		rxn							= d_particles[0].rxn;
		rn_bank						= d_particles[0].rn_bank;
		yield						= d_particles[0].yield;
		weight						= d_particles[0].weight;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	//remap to active
	int tid				=	d_remap[starting_index + tid_in];
	unsigned this_rxn 	=	rxn[    starting_index + tid_in];

	// print and return if wrong
	if ( this_rxn != 818 & this_rxn != 819 & this_rxn != 820 & this_rxn != 821 ){printf("fission kernel accessing wrong reaction @ dex %u dex_in %u rxn %u\n",tid, tid_in,this_rxn);return;} 

	// load history data
	unsigned	this_dex		=	index[  tid];
	unsigned	rn				=	rn_bank[tid];
	float		this_weight		=	weight[ tid];

	// local variables, load nu from scattering dist variables
	if (dist_scatter[this_dex].lower==0x0){
		printf("scatter pointer for rxn %d is null!\n",this_rxn);
	}
	float		nu			=	dist_scatter[this_dex].lower[0].erg;  // lower erg is nu_t, upper erg is nu_p
	unsigned	inu			=	0;
	unsigned	this_yield	=	0;

	// check nu
	if (nu==0.0){
		nu=2.8;
		printf("something is wrong with fission yields, nu = %6.4E, guessing %4.2f, rxn %u\n",0.0,nu,this_rxn); 
	}

	//  multiply nu by weight
	nu = this_weight * nu;

	// get integer part
	inu = (unsigned) nu;
	
	// sample floor or ceil based on fractional part
	if((float)inu+get_rand(&rn) <= nu){
		this_yield = inu+1;
	}
	else{
		this_yield = inu;
	}

	// put in 900 block to terminate on next sort
 	this_rxn += 100;

	printf("tid %d rxn %u wgt %6.4E yield %u\n", tid, this_rxn, this_weight, this_yield);

	// write 
	yield[  tid]					=	this_yield;
	rn_bank[tid]					=	rn;  
	rxn[starting_index + tid_in]	=	this_rxn;

}

void fission( hipStream_t stream, unsigned NUM_THREADS, unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;
	
	fission_kernel <<< blks, NUM_THREADS , 0 , stream >>> ( N, starting_index, d_xsdata, d_particles, d_remap );
	hipDeviceSynchronize();

}

