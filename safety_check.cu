#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "warp_device.cuh"
#include "check_cuda.h"
#include "wfloat3.h"

__global__ void safety_check_kernel(unsigned N, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){   

	// declare shared variables
	__shared__ 	unsigned			n_isotopes;				
	__shared__ 	unsigned			energy_grid_len;		
	__shared__ 	unsigned			total_reaction_channels;
	__shared__ 	unsigned*			rxn_numbers;			
	__shared__ 	unsigned*			rxn_numbers_total;		
	__shared__ 	float*				energy_grid;			
	__shared__ 	float*				rxn_Q;						
	__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 
	__shared__	spatial_data*		space;	
	__shared__	unsigned*			rxn;	
	__shared__	float*				E;		
	__shared__	float*				Q;		
	__shared__	unsigned*			rn_bank;
	__shared__	unsigned*			cellnum;
	__shared__	unsigned*			matnum;	
	__shared__	unsigned*			isonum;	
	__shared__	unsigned*			yield;	
	__shared__	float*				weight;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		n_isotopes					= d_xsdata[0].n_isotopes;								
		energy_grid_len				= d_xsdata[0].energy_grid_len;				
		total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		energy_grid 				= d_xsdata[0].energy_grid;						
		rxn_Q 						= d_xsdata[0].Q;												
		xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy; 
		space						= d_particles[0].space;
		rxn							= d_particles[0].rxn;
		E							= d_particles[0].E;
		Q							= d_particles[0].Q;	
		rn_bank						= d_particles[0].rn_bank;
		cellnum						= d_particles[0].cellnum;
		matnum						= d_particles[0].matnum;
		isonum						= d_particles[0].isonum;
		yield						= d_particles[0].yield;
		weight						= d_particles[0].weight;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;} 

	//remap to active
	int tid				=	d_remap[starting_index + tid_in];
	unsigned this_rxn 	=	rxn[    starting_index + tid_in];

	// check energy
	float 	this_E = E[tid];
	if (!isfinite(this_E) | this_E < 0.0){
		printf("INVALID ENERGY, tid %u tid_in %u rxn %u, E % 6.4E\n",tid,tid_in,this_rxn,this_E);
	}

	// check directions
	wfloat3		hats(space[tid].xhat,space[tid].yhat,space[tid].zhat);
	if (!isfinite(hats.x+hats.y+hats.z)){
		printf("INVALID DIRECTIONS, tid %u tid_in %u rxn %u, xhat % 6.4E yhat % 6.4E zhat % 6.4E\n",tid,tid_in,this_rxn,hats.x,hats.y,hats.z);
	}

	// check position
	wfloat3		pos(space[tid].x,space[tid].y,space[tid].z);
	if (!isfinite(pos.x+pos.y+pos.z)){
		printf("INVALID POSITIONS, tid %u tid_in %u rxn %u, x % 6.4E y % 6.4E z % 6.4E\n",tid,tid_in,this_rxn,pos.x,pos.y,pos.z);
	}

}

void safety_check(unsigned NUM_THREADS, unsigned N, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	safety_check_kernel <<< blks, NUM_THREADS >>> (  N, d_xsdata, d_particles, d_remap);
	check_cuda(hipDeviceSynchronize());

}

