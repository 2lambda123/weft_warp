#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"


_global__ void safety_check_kernel(unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){   

	// declare shared variables
	__shared__ 	unsigned			n_isotopes;				
	__shared__ 	unsigned			energy_grid_len;		
	__shared__ 	unsigned			total_reaction_channels;
	__shared__ 	unsigned*			rxn_numbers;			
	__shared__ 	unsigned*			rxn_numbers_total;		
	__shared__ 	float*				energy_grid;			
	__shared__ 	float*				rxn_Q;						
	__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 
	__shared__	spatial_data*		space;	
	__shared__	unsigned*			rxn;	
	__shared__	float*				E;		
	__shared__	float*				Q;		
	__shared__	unsigned*			rn_bank;
	__shared__	unsigned*			cellnum;
	__shared__	unsigned*			matnum;	
	__shared__	unsigned*			isonum;	
	__shared__	unsigned*			yield;	
	__shared__	float*				weight;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		n_isotopes					= d_xsdata[0].n_isotopes;								
		energy_grid_len				= d_xsdata[0].energy_grid_len;				
		total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		energy_grid 				= d_xsdata[0].energy_grid;						
		rxn_Q 						= d_xsdata[0].Q;												
		xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy; 
		space						= d_particles[0].space;
		rxn							= d_particles[0].rxn;
		E							= d_particles[0].E;
		Q							= d_particles[0].Q;	
		rn_bank						= d_particles[0].rn_bank;
		cellnum						= d_particles[0].cellnum;
		matnum						= d_particles[0].matnum;
		isonum						= d_particles[0].isonum;
		yield						= d_particles[0].yield;
		weight						= d_particles[0].weight;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;} 

	//remap to active
	int tid				=	d_remap[starting_index + tid_in];
	unsigned this_rxn 	=	rxn[    starting_index + tid_in];

	// check energy
	float this_E = E[tid];
	if (!isfinite(this_E) | this_E < 0.0){
		printf("INVALID ENERGY % 6.4E   tid %u tid_in %u rxn %u\n", this_E,tid,tid_in,this_rxn);
	}

	


}

void safety_check(unsigned NUM_THREADS, unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	safety_check_kernel <<< blks, NUM_THREADS >>> (  N, starting_index, d_xsdata, d_particles, d_remap);
	check_cuda(hipDeviceSynchronize());

}

