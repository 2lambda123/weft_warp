#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"
#include "wfloat3.h"
#include "warp_device.cuh"
#include "check_cuda.h"


__global__ void scatter_conti_kernel(unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	// declare shared variables
	//__shared__ 	unsigned			n_isotopes;				
	//__shared__ 	unsigned			energy_grid_len;		
	//__shared__ 	unsigned			total_reaction_channels;
	//__shared__ 	unsigned*			rxn_numbers;			
	//__shared__ 	unsigned*			rxn_numbers_total;		
	//__shared__ 	float*				energy_grid;			
	//__shared__ 	float*				rxn_Q;						
	//__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	//__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 
	__shared__	spatial_data*		space;	
	__shared__	unsigned*			rxn;	
	__shared__	float*				E;		
	//__shared__	float*				Q;		
	__shared__	unsigned*			rn_bank;
	//__shared__	unsigned*			cellnum;
	//__shared__	unsigned*			matnum;	
	__shared__	unsigned*			isonum;	
	//__shared__	unsigned*			yield;	
	//__shared__	float*				weight;	
	__shared__	unsigned*			index;	

	// have thread 0 of block copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		//n_isotopes					= d_xsdata[0].n_isotopes;								
		//energy_grid_len				= d_xsdata[0].energy_grid_len;				
		//total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		//rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		//rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		//energy_grid 				= d_xsdata[0].energy_grid;						
		//rxn_Q 						= d_xsdata[0].Q;												
		//xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		//temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy; 
		space						= d_particles[0].space;
		rxn							= d_particles[0].rxn;
		E							= d_particles[0].E;
		//Q							= d_particles[0].Q;	
		rn_bank						= d_particles[0].rn_bank;
		//cellnum						= d_particles[0].cellnum;
		//matnum						= d_particles[0].matnum;
		isonum						= d_particles[0].isonum;
		//yield						= d_particles[0].yield;
		//weight						= d_particles[0].weight;
		index						= d_particles[0].index;
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	// return immediately if out of bounds
	int tid_in = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid_in >= N){return;}

	//remap to active
	int tid				=	d_remap[starting_index + tid_in];
	unsigned this_rxn 	=	rxn[    starting_index + tid_in];

	// print and return if wrong
	if ( this_rxn!=91 ){printf("level scattering kernel accessing wrong reaction @ dex %u rxn %u\n",tid, this_rxn);return;} 

	// check E data pointers
	if(dist_energy == 0x0){
		printf("null pointer, energy array in continuum scatter!,tid %u rxn %u\n",tid,this_rxn);
		return;
	}

	//constants
	//const float  	pi			=   3.14159265359;
	const float  	m_n			=   1.00866491600;		// u
	//const float		kb			=	8.617332478e-11;	// MeV/k

	// load history data
	wfloat3		hats_old(space[tid].xhat,space[tid].yhat,space[tid].zhat);
	unsigned	this_tope		=	isonum[  tid];
	unsigned	this_dex		=	index[   tid];
	float		this_E			=	E[       tid];
	//float		this_Q			=	Q[       tid];
	unsigned	rn				=	rn_bank[ tid];
	float		this_awr		=	awr[ this_tope];
	//float		this_temp		=	temp[this_tope];

	// pick upper or lower via stochastic mixing
	dist_data	this_edist, this_sdist;
	dist_data	sdist_lower	=	dist_scatter[this_dex].lower[0];
	dist_data	sdist_upper	=	dist_scatter[this_dex].upper[0];
	dist_data	edist_lower	=	dist_energy[ this_dex].lower[0];
	dist_data	edist_upper	=	dist_energy[ this_dex].upper[0];
	unsigned	this_law;
	float		f			=	(this_E - edist_lower.erg) / (edist_upper.erg - edist_lower.erg);
	if( get_rand(&rn)>f ){
		this_edist	=	edist_lower;
		this_sdist	=	sdist_lower;
	}
	else{
		this_edist = edist_upper;
		this_sdist = sdist_upper;
	}
	this_law	=	this_edist.law;

	// internal kernel variables
	float  		E_target     		=   0;
	float 		speed_target     	=   sqrtf(2.0*E_target/(this_awr*m_n));
	float  		speed_n          	=   sqrtf(2.0*this_E/m_n);
	float 		E_new				=   0.0;
	float 		sampled_E			=	0.0;
	wfloat3 	v_n_cm, v_t_cm, v_n_lf, v_t_lf, v_cm, hats_new, hats_target, rotation_hat;
	float 		mu, E0, A, R;
	unsigned 	dist_index[1];    // must be declared this way in order to write to passed pointer, why??

	// ensure normalization
	hats_old = hats_old / hats_old.norm2();

	// make speed vectors, assume high enough energy to approximate target as stationary
	v_n_lf = hats_old    * speed_n;
	v_t_lf = hats_target * 0.0;

	// calculate  v_cm
	v_cm = (v_n_lf + (v_t_lf*this_awr))/(1.0+this_awr);

	//transform neutron velocity into CM frame
	v_n_cm = v_n_lf - v_cm;
	v_t_cm = v_t_lf - v_cm;


	if (this_law ==4 ){

		// sample continuous tabular
		E0 = sample_continuous_tablular( 	this_edist.len , 
											this_edist.intt , 
											get_rand(&rn) , 
											this_edist.var , 
											this_edist.pdf, 
											this_edist.cdf );
		//scale it to bins 
		sampled_E = scale_to_bins(	f, E0, 
									 this_edist.var[0],  this_edist.var[ this_edist.len-1], 
									edist_lower.var[0], edist_lower.var[edist_lower.len-1], 
									edist_upper.var[0], edist_upper.var[edist_upper.len-1] );

		// sample mu isotropically
		mu  = 2.0*get_rand(&rn)-1.0;

	}
//	else if (law==9){   //evaopration spectrum
//
//		// get tabulated temperature
//		float t0 = this_Earray[ offset              ];
//		float t1 = this_Earray[ offset + 1          ];
//		float U  = this_Earray[ offset + vlen       ];
//			  e0 = this_Earray[ offset + vlen*2     ];
//			  e1 = this_Earray[ offset + vlen*2 + 1 ];
//		float  T = 0.0;
//		float  m = 0.0;
//
//		// interpolate T
//		if (e1==e0){  // in top bin, both values are the same
//			T = t0;
//		}
//		else if (intt==2){// lin-lin interpolation
//			m = (this_E - e0)/(e1 - e0);
//			T = (1.0 - m)*t0 + m*t1;
//		}
//		else if(intt==1){// histogram interpolation
//			T  = (t1 - t0)/(e1 - e0) * this_E + t0;
//		}
//
//		// rejection sample
//		m  = (this_E - U)/T;
//		e0 = 1.0-expf(-m);
//		float x  = -logf(1.0-e0*get_rand(&rn)) - logf(1.0-e0*get_rand(&rn));
//		while (  x>m ) {
//			x  = -logf(1.0-e0*get_rand(&rn)) - logf(1.0-e0*get_rand(&rn));
//		}
//
//		// mcnp5 volIII pg 2-43
//		sampled_E = T * x;
//
//		//isotropic mu
//		if (this_Sarray==0x0){
//			mu  = 2.0*get_rand(&rn)-1.0;
//		}
//		else{
//			printf("law 9 in cscatter has angular tables\n");
//		}
//
//	}
	else if (this_law==44){

		// make sure scatter array is present
		if(dist_scatter == 0x0){
			printf("null pointer, scatter array in continuum !,dex %u rxn %u tope %u E %6.4E \n",this_dex,this_rxn,this_tope,this_E);
			return;
		}

		// correct if below lower energy?
		//if(this_E<this_edist.var[this_edist.len-1]){
		//	printf("above last e! \n");
		//	//this_E = last_E;
		//}

		// compute interpolation factor
		if(f<0){
			printf("DATA NOT WITHIN ENERGY INTERVAL tid %u rxn %u\n",tid,this_rxn);
		}

		// sample tabular on energy, but get index as well as value
		E0 = sample_continuous_tablular( 	dist_index ,
											this_edist.len , 
											2 , 
											get_rand(&rn) , 
											this_edist.var , 
											this_edist.pdf, 
											this_edist.cdf );
		//scale it to bins 
		sampled_E = scale_to_bins(	f, E0, 
									 this_edist.var[0],  this_edist.var[ this_edist.len-1], 
									edist_lower.var[0], edist_lower.var[edist_lower.len-1], 
									edist_upper.var[0], edist_upper.var[edist_upper.len-1] );

		// find correlated mu
		if (this_sdist.intt==1){
			A	=	this_sdist.var[dist_index[0]];
			R	=	this_sdist.cdf[dist_index[0]];
		}
		else if (this_sdist.intt==2){
			A	=	interpolate_linear_energy(	E0,
												this_edist.var[dist_index[0]],
												this_edist.var[dist_index[0]+1],
												this_sdist.var[dist_index[0]],
												this_sdist.var[dist_index[0]+1]);
			R	=	interpolate_linear_energy(	E0,
												this_edist.var[dist_index[0]],
												this_edist.var[dist_index[0]+1],
												this_sdist.cdf[dist_index[0]],
												this_sdist.cdf[dist_index[0]+1]);
		}
		else{
			printf("INTT=%u NOT HANDLED in law %u of continuum scatter!",this_sdist.law,this_sdist.intt);
		}
		float rn1 	= get_rand(&rn);
		if( get_rand(&rn)>R ){
			float T = (2.0*rn1-1.0)*sinhf(A);
			mu		= logf(T+sqrtf(T*T+1.0))/A;
		}
		else{
			mu		= logf(rn1*expf(A)+(1.0-rn1)*expf(-A))/A;
		}

	}
//	else if (law==61){
//
//		unsigned distloc, vloc;
//		float r = (this_E-last_E)/(next_E-last_E);
//		last_e_start = this_Earray[ offset ];
//		last_e_end   = this_Earray[ offset + vlen - 1 ];
//		next_e_start = this_Earray[ offset + 3*vlen ];
//		next_e_end   = this_Earray[ offset + 3*vlen + next_vlen - 1];
//	
//		rn1 = get_rand(&rn);
//		rn2 = get_rand(&rn);
//	
//		//sample energy dist
//		sampled_E = 0.0;
//		if(  rn2 >= r ){   //sample last E
//			distloc = 1;   // use the first flattened array
//			diff = next_e_end - next_e_start;
//			e_start = next_e_start;
//			for ( n=0 ; n<vlen-1 ; n++ ){
//				cdf0 		= this_Earray[ (offset +   vlen ) + n+0];
//				cdf1 		= this_Earray[ (offset +   vlen ) + n+1];
//				pdf0		= this_Earray[ (offset + 2*vlen ) + n+0];
//				pdf1		= this_Earray[ (offset + 2*vlen ) + n+1];
//				e0  		= this_Earray[ (offset          ) + n+0];
//				e1  		= this_Earray[ (offset          ) + n+1]; 
//				if( rn1 >= cdf0 & rn1 < cdf1 ){
//					break;
//				}
//			}
//		}
//		else{
//			distloc = this_Sarray[0];   // get location of the next flattened array
//			diff = next_e_end - next_e_start;
//			e_start = next_e_start;
//			for ( n=0 ; n<next_vlen-1 ; n++ ){
//				cdf0 		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+0];
//				cdf1  		= this_Earray[ (offset + 3*vlen +   next_vlen ) + n+1];
//				pdf0		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+0];
//				pdf1		= this_Earray[ (offset + 3*vlen + 2*next_vlen ) + n+1];
//				e0   		= this_Earray[ (offset + 3*vlen               ) + n+0];
//				e1   		= this_Earray[ (offset + 3*vlen               ) + n+1];
//				if( rn1 >= cdf0 & rn1 < cdf1 ){
//					break;
//				}
//			}
//		}
//	
//		if (intt==2){// lin-lin interpolation
//			float m 	= (pdf1 - pdf0)/(e1-e0);
//			float arg = pdf0*pdf0 + 2.0 * m * (rn1-cdf0);
//			if(arg<0){
//				E0 = e0 + (e1-e0)/(cdf1-cdf0)*(rn1-cdf0);
//			}
//			else{
//				E0 	= e0 + (  sqrtf( arg ) - pdf0) / m ;
//			}
//		}
//		else if(intt==1){// histogram interpolation
//			E0 = e0 + (rn1-cdf0)/pdf0;
//		}
//		
//		//scale it
//		E1 = last_e_start + r*( next_e_start - last_e_start );
//		Ek = last_e_end   + r*( next_e_end   - last_e_end   );
//		sampled_E = E1 +(E0-e_start)*(Ek-E1)/diff;
//
//		//
//		// sample mu from tabular distributions
//		//
//
//		// get parameters
//		unsigned vlen_S ;
//		if(distloc){
//			unsigned l = this_Sarray[0];
//			vloc   = this_Sarray[l + n] + (l + next_vlen) ; // get appropriate vector location for this E_out
//			}                
//		else{   
//			vloc   = this_Sarray[1 + n] + (1 + vlen) ;     
//		}
//		vlen_S = this_Sarray[vloc + 0];        // vector length
//		intt   = this_Sarray[vloc + 1];        // interpolation type
//		//printf("distloc %u vloc %u vlen_S %u intt %u \n",distloc,vloc,vlen_S,intt);
//
//		// sample the dist
//		rn1 = get_rand(&rn);
//		for ( n=0 ; n<vlen-1 ; n++ ){
//			cdf0 		= this_Sarray[ (vloc + 2 +   vlen_S ) + n+0];
//			cdf1  		= this_Sarray[ (vloc + 2 +   vlen_S ) + n+1];
//			pdf0		= this_Sarray[ (vloc + 2 + 2*vlen_S ) + n+0];
//			pdf1		= this_Sarray[ (vloc + 2 + 2*vlen_S ) + n+1];
//			e0   		= this_Sarray[ (vloc + 2            ) + n+0];
//			e1   		= this_Sarray[ (vloc + 2            ) + n+1];
//			if( rn1 >= cdf0 & rn1 < cdf1 ){
//				break;
//			}
//		}
//
//		// interpolate
//		if (e1==e0){  // in top bin, both values are the same
//				mu = e0;
//			}
//		else if (intt==2){// lin-lin interpolation
//			r = (rn1 - cdf0)/(cdf1 - cdf0);
//           mu = (1.0 - r)*e0 + r*e1;
//		}
//		else if(intt==1){// histogram interpolation
//			mu  = (e1 - e0)/(cdf1 - cdf0) * rn1 + e0;
//		}
//		else{
//			printf("intt in law 61 in cscatter is invlaid (%u)!\n",intt);
//		}
//		
//
//	}
	else{

		printf("LAW %u NOT HANDLED IN CONTINUUM SCATTER!  rxn %u\n",this_law,this_rxn);

	}

	// check errors
	if (!isfinite(sampled_E) | sampled_E < 0.0){
		printf("continuum scatter mis-sampled tid %i data_dex %u E %6.4E dist len %u dist_index %u... \n",tid_in,tid,sampled_E,this_sdist.len,dist_index[0]);
	}
	if (!isfinite(mu) | mu < -1.0 | mu > 1.0){
		printf("continuum scatter mis-sampled tid %i data_dex %u mu %6.4E dist len %u dist_index %u... \n",tid_in,tid,mu,this_sdist.len,dist_index[0]);
	}


	// rotate direction vector
	hats_old = v_n_cm / v_n_cm.norm2();
	hats_old = hats_old.rotate(mu, get_rand(&rn));

	//  scale to sampled energy
	v_n_cm = hats_old * sqrtf(2.0*sampled_E/m_n);
	
	// transform back to L
	v_n_lf = v_n_cm + v_cm;
	hats_new = v_n_lf / v_n_lf.norm2();
	hats_new = hats_new / hats_new.norm2(); // get higher precision, make SURE vector is length one
	
	// calculate energy in lab frame
	E_new = 0.5 * m_n * v_n_lf.dot(v_n_lf);

	//printf("tid %d law %u sampled_E %6.4E mu %6.4E\n",tid,this_law,sampled_E,mu);
	
	// write universal results
	E[tid]			=	E_new;
	space[tid].xhat	=	hats_new.x;
	space[tid].yhat	=	hats_new.y;
	space[tid].zhat	=	hats_new.z;
	rn_bank[tid]	=	rn;

}

void scatter_conti( hipStream_t stream, unsigned NUM_THREADS, unsigned N, unsigned starting_index, cross_section_data* d_xsdata, particle_data* d_particles, unsigned* d_remap){

	if(N<1){return;}
	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;
	
	scatter_conti_kernel <<< blks, NUM_THREADS , 0 , stream >>> ( N, starting_index, d_xsdata, d_particles, d_remap );
	check_cuda(hipDeviceSynchronize());

}

