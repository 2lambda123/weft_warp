#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void copy_points_kernel( unsigned Nout, unsigned * Nvalid , unsigned current_index , unsigned * to_valid, spatial_data * positions_out , spatial_data * positions_in, float*E_out, float*E_in  ){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= Nvalid[0]){return;}

	unsigned index_in  = to_valid[tid];
	unsigned index_out = current_index + tid;
	if (index_out>=Nout){index_out=index_out-Nout;} //wrap to start
	//printf("index out = %u\n",index_out);

	// copy points
	positions_out[index_out].x    			= positions_in[index_in].x; 
	positions_out[index_out].y    			= positions_in[index_in].y; 
	positions_out[index_out].z    			= positions_in[index_in].z; 
	positions_out[index_out].xhat 			= positions_in[index_in].xhat; 
	positions_out[index_out].yhat 			= positions_in[index_in].yhat; 
	positions_out[index_out].zhat 			= positions_in[index_in].zhat;
	positions_out[index_out].enforce_BC 	= positions_in[index_in].enforce_BC;
	positions_out[index_out].surf_dist 		= positions_in[index_in].surf_dist ;
	positions_out[index_out].macro_t 		= positions_in[index_in].macro_t ;
	E_out[index_out] 						= E_in[index_in];

	//printf("good point %6.4E %6.4E %6.4E\n",positions_out[index_out].x,positions_out[index_out].y,positions_out[index_out].z);


}

void copy_points( unsigned NUM_THREADS,  unsigned Nout , unsigned * Nvalid,  unsigned current_index , unsigned * to_valid , spatial_data * positions_out , spatial_data * positions_in, float*E_out, float*E_in){

	unsigned blks = ( Nout + NUM_THREADS - 1 ) / NUM_THREADS;

	copy_points_kernel <<< blks, NUM_THREADS >>> (  Nout , Nvalid,  current_index , to_valid , positions_out , positions_in , E_out, E_in);
	hipDeviceSynchronize();

}

