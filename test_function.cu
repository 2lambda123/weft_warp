#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void test_kernel( unsigned N , cross_section_data* d_xsdata, particle_data* d_particles, tally_data* d_tally, unsigned* d_remap){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}

	// declare shared variables
	__shared__ 	unsigned			n_isotopes;				
	__shared__ 	unsigned			energy_grid_len;		
	__shared__ 	unsigned			total_reaction_channels;
	__shared__ 	unsigned*			rxn_numbers;			
	__shared__ 	unsigned*			rxn_numbers_total;		
	__shared__ 	float*				energy_grid;			
	__shared__ 	float*				Q;						
	__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 

	// have thread 1 copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		n_isotopes					= d_xsdata[0].n_isotopes;								
		energy_grid_len				= d_xsdata[0].energy_grid_len;				
		total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		energy_grid 				= d_xsdata[0].energy_grid;						
		Q 							= d_xsdata[0].Q;												
		xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy;  
	}

	// make sure shared loads happen before anything else
	__syncthreads();

	// go about your thready business
	unsigned row = energy_grid_len*0.99;
	unsigned total_cols = n_isotopes + total_reaction_channels;
	unsigned this_isotope = 0;
	unsigned col_start= n_isotopes + rxn_numbers_total[this_isotope];
	unsigned col_end  = n_isotopes + rxn_numbers_total[this_isotope+1];
	unsigned col = col_start + 3;
	unsigned this_index = row*total_cols+col;

	//
	printf("\n ---- CUDA TEST FUNCTION ----\n");
	printf("\n");
	printf("tid %d here isotopes %u this isotope %u\n",tid,n_isotopes,this_isotope);
	printf("remap[%d]=%u\n",tid,d_remap[tid]);
	printf("energy of grid index %u is %10.8E\n",row,energy_grid[row]);
	printf("col start %u end %u\n",col_start,col_end);
	printf("column is %u, rxn is %u, total columns %u, index is %u, total xs is %10.8E\n",col,rxn_numbers[col],total_cols,this_index,xs[this_index]);
	printf("Q %6.4E\n",Q[col]);
	printf("awr %6.4E\n",  awr[this_isotope]);
	printf("temp %6.4E\n",temp[this_isotope]);

	//
	printf("\n");
	printf("scattering dist pointer %p\n",dist_scatter);
	printf("scattering dist pointers, lower %p upper %p\n",dist_scatter[this_index].lower,dist_scatter[this_index].upper);
	if (dist_scatter[this_index].lower != 0x0){
		printf("lower scattering dist, erg %6.8E len %u law %u intt %u\n",dist_scatter[this_index].lower[0].erg,dist_scatter[this_index].lower[0].len,dist_scatter[this_index].lower[0].law,dist_scatter[this_index].lower[0].intt);
		printf("upper scattering dist, erg %6.8E len %u law %u intt %u\n",dist_scatter[this_index].upper[0].erg,dist_scatter[this_index].upper[0].len,dist_scatter[this_index].upper[0].law,dist_scatter[this_index].upper[0].intt);
	}
	else{
		printf("Null scatter dist pointers\n");
	}

	//
	printf("\n");
	printf("energy dist pointer %p\n",dist_energy);
	printf("energy dist pointers, lower %p upper %p\n",dist_energy[this_index].lower,dist_energy[this_index].upper);
	if (dist_energy[this_index].lower != 0x0){
		printf("lower energy dist, erg %6.8E len %u law %u intt %u\n",dist_energy[this_index].lower[0].erg,dist_energy[this_index].lower[0].len,dist_energy[this_index].lower[0].law,dist_energy[this_index].lower[0].intt);
		printf("upper energy dist, erg %6.8E len %u law %u intt %u\n",dist_energy[this_index].upper[0].erg,dist_energy[this_index].upper[0].len,dist_energy[this_index].upper[0].law,dist_energy[this_index].upper[0].intt);
	}
	else{
		printf("Null energy dist pointers\n");
	}

	//
	printf("\n");


}

void test_function( unsigned NUM_THREADS,  unsigned N , cross_section_data* d_xsdata, particle_data* d_particles, tally_data* d_tally, unsigned* d_remap){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	test_kernel <<< blks, NUM_THREADS >>> (  N, d_xsdata, d_particles, d_tally , d_remap);
	hipDeviceSynchronize();

}

