#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "datadef.h"

__global__ void test_kernel( unsigned N , cross_section_data* d_xsdata, particle_data* d_particles, tally_data* d_tally){

	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if (tid >= N){return;}

	// declare shared variables
	__shared__ 	unsigned			n_isotopes;				
	__shared__ 	unsigned			energy_grid_len;		
	__shared__ 	unsigned			total_reaction_channels;
	__shared__ 	unsigned*			rxn_numbers;			
	__shared__ 	unsigned*			rxn_numbers_total;		
	__shared__ 	float*				energy_grid;			
	__shared__ 	float*				Q;						
	__shared__ 	float*				xs;						
	__shared__ 	float*				awr;					
	__shared__ 	float*				temp;					
	__shared__ 	dist_container*		dist_scatter;			
	__shared__ 	dist_container*		dist_energy; 

	// have thread 1 copy all pointers and static info into shared memory
	if (threadIdx.x == 0){
		n_isotopes					= d_xsdata[0].n_isotopes;								
		energy_grid_len				= d_xsdata[0].energy_grid_len;				
		total_reaction_channels		= d_xsdata[0].total_reaction_channels;
		rxn_numbers 				= d_xsdata[0].rxn_numbers;						
		rxn_numbers_total			= d_xsdata[0].rxn_numbers_total;					
		energy_grid 				= d_xsdata[0].energy_grid;						
		Q 							= d_xsdata[0].Q;												
		xs 							= d_xsdata[0].xs;												
		awr 						= d_xsdata[0].awr;										
		temp 						= d_xsdata[0].temp;										
		dist_scatter 				= d_xsdata[0].dist_scatter;						
		dist_energy 				= d_xsdata[0].dist_energy;  
	}

	// go about your thready business
	unsigned row = energy_grid_len*0.99;
	unsigned total_cols = n_isotopes + total_reaction_channels;
	unsigned this_isotope = 2;
	unsigned col_start= n_isotopes + rxn_numbers_total[this_isotope];
	unsigned col_end  = n_isotopes + rxn_numbers_total[this_isotope+1];
	unsigned col = col_start + 10;
	unsigned this_index = row*total_cols+col;

	//
	printf("\n");
	printf("tid %d here isotopes %u this isotope %u\n",tid,n_isotopes,this_isotope);
	printf("energy of grid index %u is %10.8E\n",row,energy_grid[row]);
	printf("column is %u, rxn is %u, total columns %u, index is %u, total xs is %10.8E\n",col,rxn_numbers[col],total_cols,this_index,xs[this_index]);
	printf("Q %6.4E\n",Q[col]);
	printf("awr %6.4E\n",  awr[this_isotope]);
	printf("temp %6.4E\n",temp[this_isotope]);
	printf("scattering dist pointer %p\n",dist_scatter);
	printf("scattering dist pointers, lower %p upper %p\n",dist_scatter[this_index].lower,dist_scatter[this_index].upper);
	if (dist_scatter[this_index].lower != 0x0){
		printf("lower scattering dist, erg %6.8E len %u law %u intt %u\n",dist_scatter[this_index].lower[0].erg,dist_scatter[this_index].lower[0].len,dist_scatter[this_index].lower[0].law,dist_scatter[this_index].lower[0].intt);
		printf("upper scattering dist, erg %6.8E len %u law %u intt %u\n",dist_scatter[this_index].upper[0].erg,dist_scatter[this_index].upper[0].len,dist_scatter[this_index].upper[0].law,dist_scatter[this_index].upper[0].intt);
	}
	else{
		printf("Null dist pointers\n");
	}


}

void test_function( unsigned NUM_THREADS,  unsigned N , cross_section_data* d_xsdata, particle_data* d_particles, tally_data* d_tally){

	unsigned blks = ( N + NUM_THREADS - 1 ) / NUM_THREADS;

	test_kernel <<< blks, NUM_THREADS >>> (  N, d_xsdata, d_particles, d_tally );
	hipDeviceSynchronize();

}

